#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "CosiftStitcher.h"
#include "opencv2/opencv.hpp"
#include "opencv2/highgui/highgui.hpp"    
#include "opencv2/nonfree/nonfree.hpp"    
#include "opencv2/legacy/legacy.hpp"  

#include <iostream>
#include <fstream>
#include <string>
#include <time.h>
#include <Windows.h>

hipError_t gCudaStatus;

#define CUDA_CHECK_CALL(fun, err_msg, return_code)					\
	gCudaStatus = fun;												\
	if(gCudaStatus != hipSuccess){									\
		fprintf(stderr, "error_code%d: %s", gCudaStatus, err_msg);	\
		return return_code;											\
	}

ConstDataGPU *const_data;
__constant__ ConstDataGPU dev_const_data[100];
ImageSize pano_size_;
ImageXYMap *dev_maps_;
ImageWeight *dev_weights_;
GPUImageData *dev_imgs_;
static int image_num_;
unsigned char *dev_pano_;

#define USE_STREAM 1
#define DST_IMAGE_CHANNEL 3

static const float WEIGHT_EPS = 1e-10f;

bool StitcherPrepared = false;//ƴ��׼����־λ��ʼ��

//Function of Class MyExposureCompensator
void MyExposureCompensator::createWeightMaps(const vector<cv::Point> &corners, const vector<Mat> &images,
	const vector<Mat> &masks, vector<Mat_<float>> &ec_maps)
{
	vector<pair<Mat, uchar> > level_masks;
	for (size_t i = 0; i < masks.size(); ++i)
		level_masks.push_back(make_pair(masks[i], 255));
	createWeightMaps(corners, images, level_masks, ec_maps);
}

void MyExposureCompensator::createWeightMaps(const vector<cv::Point> &corners, const vector<Mat> &images,
	const vector<pair<Mat, uchar>> &masks, vector<Mat_<float>> &ec_maps)
{
	CV_Assert(corners.size() == images.size() && images.size() == masks.size());

	const int num_images = static_cast<int>(images.size());

	vector<Size> bl_per_imgs(num_images);
	vector<cv::Point> block_corners;
	vector<Mat> block_images;
	vector<pair<Mat, uchar> > block_masks;

	// Construct blocks for gain compensator
	for (int img_idx = 0; img_idx < num_images; ++img_idx)
	{
		Size bl_per_img((images[img_idx].cols + bl_width_ - 1) / bl_width_,
			(images[img_idx].rows + bl_height_ - 1) / bl_height_);
		int bl_width = (images[img_idx].cols + bl_per_img.width - 1) / bl_per_img.width;
		int bl_height = (images[img_idx].rows + bl_per_img.height - 1) / bl_per_img.height;
		bl_per_imgs[img_idx] = bl_per_img;
		for (int by = 0; by < bl_per_img.height; ++by)
		{
			for (int bx = 0; bx < bl_per_img.width; ++bx)
			{
				cv::Point bl_tl(bx * bl_width, by * bl_height);
				cv::Point bl_br(min(bl_tl.x + bl_width, images[img_idx].cols),
					min(bl_tl.y + bl_height, images[img_idx].rows));

				block_corners.push_back(corners[img_idx] + bl_tl);
				block_images.push_back(images[img_idx](Rect(bl_tl, bl_br)));
				block_masks.push_back(make_pair(masks[img_idx].first(Rect(bl_tl, bl_br)), masks[img_idx].second));
			}
		}
	}
	//ʵ����GainCompensator��ÿ���鶼Ӧ�����油������
	GainCompensator compensator;
	compensator.feed(block_corners, block_images, block_masks);//�õ��鲹��ϵ��
	vector<double> gains = compensator.gains();//�õ�����ϵ��
	ec_maps.resize(num_images);//ȫ�ֱ���ec_maps��ʾ���п������

	Mat_<float> ker(1, 3);
	ker(0, 0) = 0.25; ker(0, 1) = 0.5; ker(0, 2) = 0.25;

	int bl_idx = 0;
	for (int img_idx = 0; img_idx < num_images; ++img_idx)
	{
		Size bl_per_img = bl_per_imgs[img_idx];
		ec_maps[img_idx].create(bl_per_img);

		for (int by = 0; by < bl_per_img.height; ++by)
			for (int bx = 0; bx < bl_per_img.width; ++bx, ++bl_idx)
				ec_maps[img_idx](by, bx) = static_cast<float>(gains[bl_idx]);

		sepFilter2D(ec_maps[img_idx], ec_maps[img_idx], CV_32F, ker, ker);
		sepFilter2D(ec_maps[img_idx], ec_maps[img_idx], CV_32F, ker, ker);
	}

	double max_ec = 1.0f;
	double max_ec_i, min_ec_i;
	for (int i = 0; i < num_images; i++)
	{
		cv::minMaxIdx(ec_maps[i], &min_ec_i, &max_ec_i);
		max_ec = std::max(max_ec, max_ec_i);
	}
	for (int i = 0; i < num_images; i++)
		ec_maps[i] = ec_maps[i] / ((float)(max_ec));
	ec_maps_ = ec_maps;
}

void MyExposureCompensator::feed(const vector<cv::Point> &corners, const vector<Mat> &images, vector<Mat> &masks)
{
	vector<pair<Mat, uchar> > level_masks;
	for (size_t i = 0; i < masks.size(); ++i)
		level_masks.push_back(make_pair(masks[i], 255));
	createWeightMaps(corners, images, level_masks, ec_maps_);
}

void MyExposureCompensator::gainMapResize(vector<Size> sizes_, vector<Mat_<float>> &ec_maps)
{
	int n = sizes_.size();
	for (int i = 0; i < n; i++)
	{
		Mat_<float> gain_map;
		resize(ec_maps[i], gain_map, sizes_[i], 0, 0, INTER_LINEAR);
		ec_maps[i] = gain_map.clone();
	}
}

void MyExposureCompensator::apply(int index, Mat &image)
{
	CV_Assert(image.type() == CV_8UC3);

	Mat_<float> gain_map;
	if (ec_maps_[index].size() == image.size())
		gain_map = ec_maps_[index];
	else
		resize(ec_maps_[index], gain_map, image.size(), 0, 0, INTER_LINEAR);

	for (int y = 0; y < image.rows; ++y)
	{
		const float* gain_row = gain_map.ptr<float>(y);
		cv::Point3_<uchar>* row = image.ptr<cv::Point3_<uchar> >(y);
		for (int x = 0; x < image.cols; ++x)
		{
			row[x].x = saturate_cast<uchar>(row[x].x * gain_row[x]);
			row[x].y = saturate_cast<uchar>(row[x].y * gain_row[x]);
			row[x].z = saturate_cast<uchar>(row[x].z * gain_row[x]);
		}
	}
}


//MyFeatherBlender of Class MyExposureCompensator
void MyFeatherBlender::createWeightMaps(Rect dst_roi, vector<cv::Point> corners, vector<Mat> &masks, vector<Mat> &weight_maps)
{
	dst_weight_map_.create(dst_roi.size(), CV_32F);
	dst_weight_map_.setTo(0);

	// Ϊÿһ��ͼƬ����weight map
	int image_num = masks.size();
	weight_maps.resize(image_num);
	for (int i = 0; i < image_num; i++)
	{
		createWeightMap(masks[i], m_sharpness_, weight_maps[i]);
		//cout << weight_maps[i].size() << endl;
		int dx = corners[i].x - dst_roi.x;
		int dy = corners[i].y - dst_roi.y;
		for (int y = 0; y < weight_maps[i].rows; ++y)
		{
			float* weight_row = weight_maps[i].ptr<float>(y);
			float* dst_weight_row = dst_weight_map_.ptr<float>(dy + y);
			for (int x = 0; x < weight_maps[i].cols; ++x)
			{
				//weight_row[x] = pow(weight_row[x], 0.1f);
				dst_weight_row[dx + x] += weight_row[x];
			}
		}
	}
	for (int i = 0; i < image_num; i++)
	{
		int dx = corners[i].x - dst_roi.x;
		int dy = corners[i].y - dst_roi.y;
		for (int y = 0; y < weight_maps[i].rows; ++y)
		{
			float* weight_row = weight_maps[i].ptr<float>(y);
			float* dst_weight_row = dst_weight_map_.ptr<float>(dy + y);
			for (int x = 0; x < weight_maps[i].cols; ++x)
				weight_row[x] = weight_row[x] / (dst_weight_row[dx + x] + WEIGHT_EPS);
		}
	}
}

void MyFeatherBlender::prepare(Rect dst_roi, vector<cv::Point> corners, vector<Mat> &masks)
{
	dst_.create(dst_roi.size(), CV_16SC3);
	dst_.setTo(Scalar::all(0));
	dst_mask_.create(dst_roi.size(), CV_8U);
	dst_mask_.setTo(Scalar::all(0));
	dst_roi_ = dst_roi;

	this->createWeightMaps(dst_roi, corners, masks, weight_maps_);
}

void MyFeatherBlender::feed(const Mat &img, const Mat &mask, cv::Point tl, int img_idx)
{
	CV_Assert(img.type() == CV_16SC3);
	CV_Assert(mask.type() == CV_8U);

	int dx = tl.x - dst_roi_.x;
	int dy = tl.y - dst_roi_.y;

	for (int y = 0; y < img.rows; ++y)
	{
		const cv::Point3_<short>* src_row = img.ptr<cv::Point3_<short> >(y);
		cv::Point3_<short>* dst_row = dst_.ptr<cv::Point3_<short> >(dy + y);
		const float* weight_row = weight_maps_[img_idx].ptr<float>(y);

		for (int x = 0; x < img.cols; ++x)
		{
			dst_row[dx + x].x += static_cast<short>(src_row[x].x * weight_row[x]);
			dst_row[dx + x].y += static_cast<short>(src_row[x].y * weight_row[x]);
			dst_row[dx + x].z += static_cast<short>(src_row[x].z * weight_row[x]);
		}
	}
}

void MyFeatherBlender::blend(Mat &dst, Mat &dst_mask)
{
	dst_mask_ = dst_weight_map_ > WEIGHT_EPS;
	dst = dst_;
	dst_mask = dst_mask_;
}


//Function of Class CosiftStitcher

#define STREAM_NUM 2

/*====================================================================
������      :ucharToMat
����        :��uchar���͵�����ת��ΪMat����
�������˵��:
����ֵ˵��  ��0 = ��ʾ�ɹ���0 >  ��ʾʧ��
ע�ͣ������ڵ��Թ۲�
******************************************************************************/
void ucharToMat(uchar *p2, Mat& src)
{
	int nr = src.rows;
	int nc = src.cols * src.channels();//ÿһ�е�Ԫ�ظ���
	for (int j = 0; j < nr; j++)
	{
		uchar* data = src.ptr<uchar>(j);
		for (int i = 0; i < nc; i++)
		{
			*data++ = *p2++;
		}
	}
}

//Kernel of Stitch
__global__ void Stitch_kernel(unsigned char *image, ImageXYMap xymap, ImageWeight weight, unsigned char *dst, int img_idx, ImageSize pano_size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if ((i < dev_const_data[img_idx].warped_width) && (j < dev_const_data[img_idx].warped_height))
	{
		int data_idx = j * dev_const_data[img_idx].warped_width + i;
		float map_x = xymap.xmap[data_idx];
		int map_x1 = (int)map_x;
		if (map_x1 >= 0)
		{
			float map_y = xymap.ymap[data_idx];
			int map_y1 = (int)map_y;
			int map_x2 = map_x1 + 1;
			int map_y2 = map_y1 + 1;

			int dst_data_idx = ((j + dev_const_data[img_idx].corner_y) * pano_size.width + i + dev_const_data[img_idx].corner_x) * DST_IMAGE_CHANNEL;

			float dx1 = map_x - map_x1;
			float dy1 = map_y - map_y1;
			float dx2 = map_x2 - map_x;
			float dy2 = map_y2 - map_y;
			int img_data_idx11 = (map_y1 * dev_const_data[img_idx].width + map_x1) * 3;
			int img_data_idx12 = (map_y2 * dev_const_data[img_idx].width + map_x1) * 3;
			int img_data_idx21 = (map_y1 * dev_const_data[img_idx].width + map_x2) * 3;
			int img_data_idx22 = (map_y2 * dev_const_data[img_idx].width + map_x2) * 3;
			float total_weight = weight.total_weight[data_idx];

			for (int channel = 0; channel < 3; channel++)
			{
				dst[dst_data_idx + channel] += (unsigned char)((
					image[img_data_idx11 + channel] * dx2 * dy2 +
					image[img_data_idx12 + channel] * dx2 * dy1 +
					image[img_data_idx21 + channel] * dx1 * dy2 +
					image[img_data_idx22 + channel] * dx1 * dy1
					) * total_weight);
			}
		}
	}
}

CosiftStitcher::CosiftStitcher(InputArray images)
{
	is_mapping = false;

	is_preview_ = true;
	is_save_video_ = true;//Save Output Stitched Vedio
	start_frame_index_ = 0;
	end_frame_index_ = -1;
	is_try_gpu_ = true;
	is_debug_ = false;
	trim_type_ = CosiftStitcher::TRIM_NO;

	work_megapix_ = 1.0;//-1;//
	seam_megapix_ = 0.2;//-1;//
	is_prepared_ = false;
	conf_thresh_ = 1.f;
	features_type_ = "orb";//"surf";//
	ba_cost_func_ = "ray";
	ba_refine_mask_ = "xxxxx";
	is_do_wave_correct_ = true;
	wave_correct_ = detail::WAVE_CORRECT_HORIZ;
	is_save_graph_ = false;
	warp_type_ = "cylindrical";//"plane";//"apap";//"paniniA2B1";//"transverseMercator";//"spherical";//
	expos_comp_type_ = ExposureCompensator::GAIN_BLOCKS;//ExposureCompensator::GAIN;//
	match_conf_ = 0.3f;
	seam_find_type_ = "gc_color";//"voronoi";//
	blend_type_ = Blender::FEATHER;//Blender::MULTI_BAND;//Blender::NO;//
	blend_strength_ = 5;

	//	��ȡ��ǰϵͳ�ĺ���
	SYSTEM_INFO sys_info;
	GetSystemInfo(&sys_info);
	parallel_num_ = sys_info.dwNumberOfProcessors;

	// ����ʱ���õ�һ֡�궨ƴ�Ӳ���

	vector<Mat> srcVec;
	images.getMatVector(srcVec);
	printf("Stitcher is preparing ");
	Prepare(srcVec);//Ĭ��ʹ��Classical�㷨
	if (is_prepared_) {
		StitcherPrepared = true;//׼�����
	}
	else {
		perror("\nError in preparation!\n");
	}
}

CosiftStitcher::~CosiftStitcher()
{
	DevFree();
}

hipError_t CosiftStitcher::DevMalloc(int num_images)
{
	hipError_t cudaStatus = hipSetDevice(0);//Ĭ��0��GPU
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return hipError_t::hipErrorLaunchFailure;
	}
	image_num_ = num_images;
	const_data = (ConstDataGPU *)(malloc(num_images * sizeof(ConstDataGPU)));
	dev_maps_ = (ImageXYMap *)(malloc(num_images * sizeof(ImageXYMap)));
	dev_weights_ = (ImageWeight *)(malloc(num_images * sizeof(ImageWeight)));
	dev_imgs_ = (GPUImageData *)(malloc(num_images * sizeof(GPUImageData)));//dev_imgs_[0].data = 0;
	return hipError_t::hipSuccess;
}

int CosiftStitcher::DevDataUpload(C2GInitData *c2g_data, int pano_height, int pano_width)
{
	for (int i = 0; i < image_num_; i++)
	{
		const_data[i].warped_height = c2g_data[i].warped_height;
		const_data[i].warped_width = c2g_data[i].warped_width;
		const_data[i].height = c2g_data[i].height;
		const_data[i].width = c2g_data[i].width;
		const_data[i].corner_x = c2g_data[i].corner_x;
		const_data[i].corner_y = c2g_data[i].corner_y;

		int xy_map_size = c2g_data[i].warped_height * c2g_data[i].warped_width * sizeof(float);
		int img_size = c2g_data[i].height * c2g_data[i].width * 3 * sizeof(unsigned char);

		//	��xmap��ymap���Դ��Ϸ���ռ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_maps_[i].xmap), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_maps_[i].ymap), xy_map_size), "hipMalloc failed!\n", -2);

		//	��Ȩ�ؾ������Դ��Ϸ���ռ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].ec_weight), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].blend_weight), xy_map_size), "hipMalloc failed!\n", -2);
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_weights_[i].total_weight), xy_map_size), "hipMalloc failed!\n", -2);

		//	��ÿһ֡ͼ������Դ�
		CUDA_CHECK_CALL(hipMalloc((void**)&(dev_imgs_[i].data), img_size), "hipMalloc failed!\n", -2);

		//	��������
		CUDA_CHECK_CALL(hipMemcpy(dev_maps_[i].xmap, c2g_data[i].xmap, xy_map_size, hipMemcpyHostToDevice),
			"hipMemcpy xmap failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_maps_[i].ymap, c2g_data[i].ymap, xy_map_size, hipMemcpyHostToDevice),
			"hipMemcpy ymap failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].ec_weight, c2g_data[i].ec_weight, xy_map_size, hipMemcpyHostToDevice),
			"hipMemcpy ec_weight failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].blend_weight, c2g_data[i].blend_weight, xy_map_size, hipMemcpyHostToDevice),
			"hipMemcpy blend_weight failed!\n", -2);
		CUDA_CHECK_CALL(hipMemcpy(dev_weights_[i].total_weight, c2g_data[i].total_weight, xy_map_size, hipMemcpyHostToDevice),
			"hipMemcpy blend_weight failed!\n", -2);
	}
	//	�����洢��
	CUDA_CHECK_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_const_data), const_data, image_num_ * sizeof(ConstDataGPU)),
		"hipMemcpyToSymbol failed\n", -2);

	pano_size_.height = pano_height;
	pano_size_.width = pano_width;
	int pano_malloc_size = pano_height * pano_width * DST_IMAGE_CHANNEL * sizeof(unsigned char);
	//	��ȫ��ͼ������Դ��Ϸ���ռ�
	CUDA_CHECK_CALL(hipMalloc((void**)&(dev_pano_), pano_malloc_size), "hipMalloc failed!\n", -2);
	return 0;
}

int CosiftStitcher::Cuda_Stitch(GPUImageData *images, unsigned char *dst)
{
	int pano_malloc_size = pano_size_.height * pano_size_.width * DST_IMAGE_CHANNEL * sizeof(unsigned char);
	CUDA_CHECK_CALL(hipMemset(dev_pano_, 0, pano_malloc_size), "hipMemset failed!\n", -2);

	for (int i = 0; i < image_num_; i++)
	{
		int img_size = const_data[i].height * const_data[i].width * 3 * sizeof(unsigned char);
		CUDA_CHECK_CALL(hipMemcpy(dev_imgs_[i].data, images[i].data, img_size, hipMemcpyHostToDevice),
			"hipMemcpy images failed\n", -2);		//	2ms/f
		dim3 dimBlock(32, 16);
		dim3 dimGrid((const_data[i].warped_width + dimBlock.x - 1) / dimBlock.x,
			(const_data[i].warped_height + dimBlock.y - 1) / dimBlock.y);
		Stitch_kernel << <dimGrid, dimBlock >> >(dev_imgs_[i].data, dev_maps_[i], dev_weights_[i], dev_pano_, i, pano_size_);		//	4.1ms/f
	}

	CUDA_CHECK_CALL(hipDeviceSynchronize(), "hipDeviceSynchronize failed!\n", -2);
	CUDA_CHECK_CALL(hipMemcpy(dst, dev_pano_, pano_malloc_size, hipMemcpyDeviceToHost),
		"hipMemcpy to dst failed\n", -2);			//	1.4ms/f
	return 0;
}

Stitcher::Status CosiftStitcher::stitch(InputArray images, OutputArray pano)
{
	vector<Mat> srcVec;
	images.getMatVector(srcVec);
	Mat &dst = pano.getMatRef();
	if (dst.empty())
		dst.create(dst_roi_.size(), CV_8UC3);

	int image_num = srcVec.size();
	GPUImageData *srcimages = new GPUImageData[image_num];
	for (int i = 0; i < image_num; i++)
		srcimages[i].data = srcVec[i].ptr<uchar>(0);

	//������
	Mat t(srcVec[0].rows, srcVec[0].cols, CV_8UC3, Scalar(0, 0, 0));
	ucharToMat(srcVec[0].data, t);
	Mat t1(srcVec[0].rows, srcVec[0].cols, CV_8UC3, Scalar(0, 0, 0));
	ucharToMat(srcVec[1].data, t1);

	//int flag = Cuda_Stitch(srcimages, dst.ptr<uchar>(0));
	StitchFrame(srcVec, dst);

	free(srcimages);
	return Stitcher::OK;
}

int CosiftStitcher::DevFree()
{
	for (int i = 0; i < image_num_; i++)
	{
		hipFree(dev_maps_[i].xmap);
		hipFree(dev_maps_[i].ymap);
		hipFree(dev_weights_[i].ec_weight);
		hipFree(dev_weights_[i].blend_weight);
		hipFree(dev_weights_[i].total_weight);
	}
	free(const_data);
	free(dev_imgs_);
	free(dev_maps_);
	free(dev_weights_);
	return 0;
}

Stitcher::Status CosiftStitcher::mapping(InputArray images, OutputArray pano)
{

	return Stitcher::OK;
}

//int CosiftStitcher::stitch(vector<VideoCapture> &captures, string &writer_file_name)
//{
//	int video_num = captures.size();
//	vector<Mat> src(video_num);
//	Mat frame, dst, show_dst;
//
//	//	Debug����Ϣ
//	bool is_save_input_frames = false;
//	bool is_save_output_frames = true;
//
//	double fps = captures[0].get(CV_CAP_PROP_FPS);
//
//	// skip some frames
//	for (int j = 0; j < video_num; j++)
//		for (int i = 0; i < start_frame_index_; i++)
//			captures[j].read(frame);
//
//	// ��һ֡����һЩ��ʼ��������ȷ�������Ƶ�ķֱ���
//	for (int j = 0; j < video_num; j++)
//	{
//		if (!captures[j].read(frame))
//			return -1;
//		frame.copyTo(src[j]);
//		if (is_debug_)
//		{
//			char img_save_name[100];
//			sprintf(img_save_name, "/%d.jpg", j + 1);
//			imwrite(debug_dir_path_ + img_save_name, src[j]);
//		}
//	}
//
//	long prepare_start_clock = clock();
//	int prepare_status = Prepare(src);
//	//	����ORB�������ԣ�����Ļ���ʹ��SURF����Ȼ�����򱨴�������Ƶ����������
//	if (prepare_status == STITCH_CONFIG_ERROR)
//	{
//		cout << "video stitch config error!" << endl;
//		return -1;
//	}
//	if (prepare_status != STITCH_SUCCESS)
//	{
//		features_type_ = "surf";
//		cout << "video stitch first try failed, second try ... " << endl;
//		if (Prepare(src) != STITCH_SUCCESS)
//		{
//			cout << "videos input are invalid. Initialization failed." << endl;
//			return -1;
//		}
//	}
//	long prepare_end_clock = clock();
//	cout << "prepare time: " << prepare_end_clock - prepare_start_clock << "ms" << endl;
//	long first_frame_stitching_start = clock();
//	StitchFrame(src, dst);
//	long first_frame_stitching_end = clock();
//	cout << "first_frame time: " << first_frame_stitching_end - first_frame_stitching_start << "ms" << endl;
//	if (is_debug_)	//�����һ֡ƴ�ӽ����mask
//	{
//		imwrite(debug_dir_path_ + "/res.jpg", dst);
//		vector<Mat> img_masks(video_num);
//		for (int i = 0; i < video_num; i++)
//		{
//			img_masks[i].create(src[i].rows, src[i].cols, CV_8UC3);
//			img_masks[i].setTo(Scalar::all(255));
//		}
//		Mat dst_mask;
//		StitchFrame(img_masks, dst_mask);
//		imwrite(debug_dir_path_ + "/mask.jpg", dst_mask);
//	}
//
//	// ���������Ƶ
//	VideoWriter writer;
//	if (is_save_video_)
//	{
//		writer.open(writer_file_name, CV_FOURCC('D', 'I', 'V', '3'), 20, Size(dst.cols, dst.rows));
//		writer.write(dst);
//	}
//
//
//	// ��ʼƴ��
//	double stitch_time = 0;
//
//	FrameInfo frame_info;
//	frame_info.src.resize(video_num);
//
//	int frameidx = 1;
//
//	cout << "Stitching..." << endl;
//
//	string window_name = "��Ƶƴ��";
//	if (is_preview_)
//		namedWindow(window_name, CV_WINDOW_NORMAL);//������Ԥ��
//	double show_scale = 1.0, scale_interval = 0.03;
//	int frame_show_interval = cvFloor(1000 / fps);
//
//	int failed_frame_count = 0;
//
//	char log_string[1000];
//	char log_file_name[200];
//	SYSTEMTIME sys_time = { 0 };
//	GetLocalTime(&sys_time);
//	sprintf(log_file_name, "%d%02d%02d-%02d%02d%02d.log",
//		sys_time.wYear, sys_time.wMonth, sys_time.wDay, sys_time.wHour, sys_time.wMinute, sys_time.wSecond);
//	ofstream log_file;
//	if (is_debug_)
//		log_file.open(debug_dir_path_ + log_file_name);
//	long long startTime = clock();
//	while (true)
//	{
//		long frame_time = 0;
//		//	�ɼ�
//		long cap_start_clock = clock();
//		int j;
//		for (j = 0; j < video_num; j++)
//		{
//			if (!captures[j].read(frame))
//				break;
//			frame.copyTo(frame_info.src[j]);
//		}
//		frame_info.frame_idx = frameidx;
//		frameidx++;
//		if (j != video_num || (end_frame_index_ >= 0 && frameidx >= end_frame_index_))	//��һ����ƵԴ��������ֹͣƴ��
//			break;
//
//		//	ƴ��
//		long stitch_start_clock = clock();
//		frame_info.stitch_status = StitchFrame(frame_info.src, frame_info.dst);
//		long stitch_clock = clock();
//		sprintf(log_string, "\tframe %d: stitch(%dms), capture(%dms)",
//			frame_info.frame_idx, stitch_clock - stitch_start_clock, stitch_start_clock - cap_start_clock);
//		printf("%s", log_string);
//		if (is_debug_)
//			log_file << log_string << endl;
//		stitch_time += stitch_clock - stitch_start_clock;
//		frame_time += stitch_clock - cap_start_clock;
//
//		//	ƴ��ʧ��
//		if (frame_info.stitch_status != 0)
//		{
//			cout << "failed\n";
//			if (is_debug_)
//				log_file << "failed" << endl;
//			failed_frame_count++;
//			break;
//		}
//
//		//	������Ƶ
//		if (is_save_video_)
//		{
//			cout << ", write(";
//			if (is_save_output_frames)
//			{
//				char img_save_name[100];
//				sprintf(img_save_name, "/images/%d.jpg", frame_info.frame_idx);
//				imwrite(debug_dir_path_ + img_save_name, frame_info.dst);
//			}
//			long write_start_clock = clock();
//			writer.write(frame_info.dst);
//			long write_clock = clock();
//			cout << write_clock - write_start_clock << "ms)";
//			frame_time += write_clock - write_start_clock;
//		}
//		cout << endl;
//
//		//	��ʾ---
//		if (is_preview_)
//		{
//			int key = waitKey(std::max(1, (int)(frame_show_interval - frame_time)));
//			if (key == 27)	//	ESC(ASCII = 27)
//				break;
//			else if (key == 61 || key == 43)	//	+
//				show_scale += scale_interval;
//			else if (key == 45)				//	-
//				if (show_scale >= scale_interval)
//					show_scale -= scale_interval;
//			resize(frame_info.dst, show_dst, Size(show_scale * dst.cols, show_scale * dst.rows));
//			imshow(window_name, show_dst);
//		}
//	}
//	long long endTime = clock();
//	cout << "test " << endTime - startTime << endl;
//	cout << "\nStitch over" << endl;
//	cout << failed_frame_count << " frames failed." << endl;
//	cout << "\tfull view angle is " << cvRound(view_angle_) << "��" << endl;
//	if (is_debug_)
//		log_file << "\tfull view angle is " << cvRound(view_angle_) << "��" << endl;
//	writer.release();
//
//	cout << "\ton average: stitch time = " << stitch_time / (frameidx - 1) << "ms" << endl;
//	cout << "\tcenter: (" << -dst_roi_.x << ", " << -dst_roi_.y << ")" << endl;
//	if (is_debug_)
//	{
//		log_file << "\ton average: stitch time = " << stitch_time / (frameidx - 1) << "ms" << endl;
//		log_file << "\tcenter: (" << -dst_roi_.x << ", " << -dst_roi_.y << ")" << endl;
//		log_file.close();
//	}
//
//	return 0;
//}

void CosiftStitcher::InitMembers(int num_images)
{
}

/*
*	��ʼͼ����ֱܷ��ʺܸߣ�����һ�����������������ʱ��Ч��
*/
void CosiftStitcher::SetScales(vector<Mat> &src)
{
	if (work_megapix_ < 0)
		work_scale_ = 1.0;
	else
		work_scale_ = min(1.0, sqrt(work_megapix_ * 1e6 / src[0].size().area()));

	if (seam_megapix_ < 0)
		seam_scale_ = 1.0;
	else
		seam_scale_ = min(1.0, sqrt(seam_megapix_ * 1e6 / src[0].size().area()));
}

/*
*	������ȡ��֧��SURF��ORB
*/
int CosiftStitcher::FindFeatures(vector<Mat> &src, vector<ImageFeatures> &features)
{
	Ptr<FeaturesFinder> finder;
	if (features_type_ == "surf")
	{
#ifdef HAVE_OPENCV_GPU
		if (is_try_gpu_ && gpu::getCudaEnabledDeviceCount() > 0)
			finder = new SurfFeaturesFinderGpu();
		else
#endif
			finder = new SurfFeaturesFinder();
	}
	else if (features_type_ == "orb")
	{
		finder = new OrbFeaturesFinder();//Size(3,1), 1500, 1.3, 5);
	}
	else
	{
		cout << "Unknown 2D features type: '" << features_type_ << "'.\n";
		return Stitcher::ERR_NEED_MORE_IMGS;
	}

	int num_images = static_cast<int>(src.size());
	Mat full_img, img;

	for (int i = 0; i < num_images; ++i)
	{
		full_img = src[i].clone();//

		if (work_megapix_ < 0)
			img = full_img;
		else
			resize(full_img, img, Size(), work_scale_, work_scale_);

		(*finder)(img, features[i]);
		//LOGLN("Features in image #" << i+1 << "("<<img.size()<< "): " << features[i].keycv::Points.size());
		features[i].img_idx = i;
	}

	finder->collectGarbage();
	full_img.release();
	img.release();

	return Stitcher::OK;
}

/*
* ����ƥ�䣬Ȼ��ȥ������ͼƬ��������ʵ��ʱ��һ����������ͼƬ������ֹ�㷨
* ����ֵ��
*		0	����	����
*		-2	����	��������ͼƬ
*/
int CosiftStitcher::MatchImages(vector<ImageFeatures> &features, vector<MatchesInfo> &pairwise_matches)
{
	int total_num_images = static_cast<int>(features.size());

	BestOf2NearestMatcher matcher(is_try_gpu_, match_conf_);
	matcher(features, pairwise_matches);
	matcher.collectGarbage();

	// ȥ������ͼ��
	vector<int> indices = leaveBiggestComponent(features, pairwise_matches, conf_thresh_);

	// һ����������ͼƬ������ֹ�㷨
	int num_images = static_cast<int>(indices.size());
	if (num_images != total_num_images)
	{
		LOGLN(total_num_images - num_images << " videos are invaild");
		return Stitcher::ERR_NEED_MORE_IMGS;
	}

	return Stitcher::OK;
}

/*
* ������궨
*/
int CosiftStitcher::CalibrateCameras(vector<ImageFeatures> &features, vector<MatchesInfo> &pairwise_matches, vector<CameraParams> &cameras)
{
	HomographyBasedEstimator estimator;
	Ptr<detail::BundleAdjusterBase> adjuster;
	Mat_<uchar> refine_mask;
	vector<double> focals;

	estimator(features, pairwise_matches, cameras);

	for (size_t i = 0; i < cameras.size(); ++i)
	{
		Mat R;
		cameras[i].R.convertTo(R, CV_32F);
		cameras[i].R = R;
		LOGLN("Initial intrinsics #" << i << ":\n" << cameras[i].K());
	}

	if (ba_cost_func_ == "reproj") adjuster = new detail::BundleAdjusterReproj();
	else if (ba_cost_func_ == "ray") adjuster = new detail::BundleAdjusterRay();
	else
	{
		cout << "Unknown bundle adjustment cost function: '" << ba_cost_func_ << "'.\n";
		return Stitcher::ERR_NEED_MORE_IMGS;
	}
	adjuster->setConfThresh(conf_thresh_);
	refine_mask = Mat::zeros(3, 3, CV_8U);
	if (ba_refine_mask_[0] == 'x') refine_mask(0, 0) = 1;
	if (ba_refine_mask_[1] == 'x') refine_mask(0, 1) = 1;
	if (ba_refine_mask_[2] == 'x') refine_mask(0, 2) = 1;
	if (ba_refine_mask_[3] == 'x') refine_mask(1, 1) = 1;
	if (ba_refine_mask_[4] == 'x') refine_mask(1, 2) = 1;
	adjuster->setRefinementMask(refine_mask);
	(*adjuster)(features, pairwise_matches, cameras);

	// Find median focal length
	for (size_t i = 0; i < cameras.size(); ++i)
	{
		focals.push_back(cameras[i].focal);
		LOGLN("Camera #" << i + 1 << ":\n" << cameras[i].t << cameras[i].R);
	}

	sort(focals.begin(), focals.end());
	if (focals.size() % 2 == 1)
		median_focal_len_ = static_cast<float>(focals[focals.size() / 2]);
	else
		median_focal_len_ = static_cast<float>(focals[focals.size() / 2 - 1] + focals[focals.size() / 2]) * 0.5f;

	if (is_do_wave_correct_)
	{
		vector<Mat> rmats;
		for (size_t i = 0; i < cameras.size(); ++i)
			rmats.push_back(cameras[i].R);
		waveCorrect(rmats, wave_correct_);
		for (size_t i = 0; i < cameras.size(); ++i)
			cameras[i].R = rmats[i];
	}

	if (is_debug_)
		this->saveCameraParam(debug_dir_path_ + "/camera_param.dat");

	return Stitcher::OK;
}

/*
*	����ˮƽ�ӽǣ������ж��Ƿ�������ƽ��ͶӰ
*/
double CosiftStitcher::GetViewAngle(vector<Mat> &src, vector<CameraParams> &cameras)
{
	Ptr<WarperCreator> warper_creator = new cv::CylindricalWarper();
	Ptr<RotationWarper> warper = warper_creator->create(median_focal_len_);

	int num_images = static_cast<int>(src.size());
	vector<cv::Point> corners;
	vector<Size> sizes;
	for (int i = 0; i < num_images; ++i)
	{
		Mat_<float> K;
		cameras[i].K().convertTo(K, CV_32F);
		Rect roi = warper->warpRoi(Size(src[i].cols * work_scale_, src[i].rows * work_scale_), K, cameras[i].R);
		corners.push_back(roi.tl());
		sizes.push_back(roi.size());
	}
	Rect result_roi = resultRoi(corners, sizes);
	double view_angle = result_roi.width * 180.0 / (median_focal_len_  * CV_PI);
	return view_angle;
}

/*
*	����ӷ�֮ǰ����Ҫ�Ȱ�ԭʼͼ���mask�����������ͶӰ
*/
int CosiftStitcher::WarpForSeam(vector<Mat> &src, vector<CameraParams> &cameras, vector<Mat> &masks_warped, vector<Mat> &images_warped)
{
	// Warp images and their masks
#ifdef HAVE_OPENCV_GPU
	if (is_try_gpu_ && gpu::getCudaEnabledDeviceCount() > 0)
	{
		if (warp_type_ == "plane") warper_creator_ = new cv::PlaneWarperGpu();
		else if (warp_type_ == "cylindrical") warper_creator_ = new cv::CylindricalWarperGpu();
		else if (warp_type_ == "spherical") warper_creator_ = new cv::SphericalWarperGpu();
	}
	else
#endif
	{
		if (warp_type_ == "plane") warper_creator_ = new cv::PlaneWarper();
		else if (warp_type_ == "cylindrical") warper_creator_ = new cv::CylindricalWarper();
		else if (warp_type_ == "spherical") warper_creator_ = new cv::SphericalWarper();
		else if (warp_type_ == "fisheye") warper_creator_ = new cv::FisheyeWarper();
		else if (warp_type_ == "stereographic") warper_creator_ = new cv::StereographicWarper();
		else if (warp_type_ == "compressedPlaneA2B1") warper_creator_ = new cv::CompressedRectilinearWarper(2, 1);
		else if (warp_type_ == "compressedPlaneA1.5B1") warper_creator_ = new cv::CompressedRectilinearWarper(1.5, 1);
		else if (warp_type_ == "compressedPlanePortraitA2B1") warper_creator_ = new cv::CompressedRectilinearPortraitWarper(2, 1);
		else if (warp_type_ == "compressedPlanePortraitA1.5B1") warper_creator_ = new cv::CompressedRectilinearPortraitWarper(1.5, 1);
		else if (warp_type_ == "paniniA2B1") warper_creator_ = new cv::PaniniWarper(2, 1);
		else if (warp_type_ == "paniniA1.5B1") warper_creator_ = new cv::PaniniWarper(1.5, 1);
		else if (warp_type_ == "paniniPortraitA2B1") warper_creator_ = new cv::PaniniPortraitWarper(2, 1);
		else if (warp_type_ == "paniniPortraitA1.5B1") warper_creator_ = new cv::PaniniPortraitWarper(1.5, 1);
		else if (warp_type_ == "mercator") warper_creator_ = new cv::MercatorWarper();
		else if (warp_type_ == "transverseMercator") warper_creator_ = new cv::TransverseMercatorWarper();
	}

	if (warper_creator_.empty())
	{
		cout << "Can't create the following warper '" << warp_type_ << "'\n";
		return Stitcher::ERR_NEED_MORE_IMGS;
	}

	float warp_scale = static_cast<float>(median_focal_len_ * seam_scale_ / work_scale_);
	Ptr<RotationWarper> warper = warper_creator_->create(warp_scale);
	int full_pano_width = cvFloor(warp_scale * 2 * CV_PI);

	int num_images = static_cast<int>(src.size());
	Mat img, mask;
	for (int i = 0; i < num_images; ++i)
	{
		Mat_<float> K;
		cameras[i].K().convertTo(K, CV_32F);
		float swa = (float)seam_scale_ / work_scale_;
		K(0, 0) *= swa; K(0, 2) *= swa;
		K(1, 1) *= swa; K(1, 2) *= swa;

		if (seam_megapix_ < 0)
			img = src[i].clone();
		else
			resize(src[i], img, Size(), seam_scale_, seam_scale_);

		mask.create(img.size(), CV_8U);
		mask.setTo(Scalar::all(255));
		Mat tmp_mask_warped, tmp_img_warped;
		cv::Point tmp_corner;
		Size tmp_size;
		warper->warp(mask, K, cameras[i].R, INTER_NEAREST, BORDER_CONSTANT, tmp_mask_warped);

		//	����360��ƴ�ӵ��������
		tmp_corner = warper->warp(img, K, cameras[i].R, INTER_LINEAR, BORDER_REFLECT, tmp_img_warped);
		//cout << "warped width = " << tmp_mask_warped.cols << ", pano width = " << full_pano_width << endl;
		if (abs(tmp_mask_warped.cols - full_pano_width) <= 10)
		{
			int x1, x2;
			FindWidestInpaintRange(tmp_mask_warped, x1, x2);
			Mat mask1, mask2, img1, img2;
			Rect rect1(0, 0, x1, tmp_mask_warped.rows), rect2(x2 + 1, 0, tmp_mask_warped.cols - 1 - x2, tmp_mask_warped.rows);
			tmp_mask_warped(rect1).copyTo(mask1);
			tmp_mask_warped(rect2).copyTo(mask2);
			masks_warped.push_back(mask1);
			masks_warped.push_back(mask2);

			tmp_img_warped(rect1).copyTo(img1);
			tmp_img_warped(rect2).copyTo(img2);
			images_warped.push_back(img1);
			images_warped.push_back(img2);

			corners_.push_back(tmp_corner);
			corners_.push_back(tmp_corner + rect2.tl());

			sizes_.push_back(rect1.size());
			sizes_.push_back(rect2.size());
		}
		else
		{
			masks_warped.push_back(tmp_mask_warped);
			corners_.push_back(tmp_corner);
			images_warped.push_back(tmp_img_warped);
			sizes_.push_back(tmp_img_warped.size());
		}

	}
	return Stitcher::OK;
}

/*
*	���360��ƴ�����⡣���ں��360��ӷ��ͼƬ���ҵ�����inpaint����[x1, x2]
*/
int CosiftStitcher::FindWidestInpaintRange(Mat mask, int &x1, int &x2)
{
	vector<int> sum_row(mask.cols);
	uchar *mask_ptr = mask.ptr<uchar>(0);
	for (int x = 0; x < mask.cols; x++)
		sum_row[x] = 0;
	for (int x = 0; x < mask.cols; x++)
		for (int y = 0; y < mask.rows; y++)
			if (mask_ptr[y * mask.cols + x] != 0)
				sum_row[x] = 1;

	int cur_x1, cur_x2, max_range = 0;
	for (int x = 1; x < mask.cols; x++)	//	����߿϶���1
	{
		if (sum_row[x - 1] == 1 && sum_row[x] == 0)
			cur_x1 = x;
		else if (sum_row[x - 1] == 0 && sum_row[x] == 1)
		{
			cur_x2 = x - 1;
			if (cur_x2 - cur_x1 > max_range)
			{
				x1 = cur_x1;
				x2 = cur_x2;
			}
		}
	}
	return 0;
}

/*
*	����ӷ�
*/
int CosiftStitcher::FindSeam(vector<Mat> &images_warped, vector<Mat> &masks_warped)
{
	int num_images = static_cast<int>(images_warped.size());
	vector<Mat> images_warped_f(num_images);
	for (int i = 0; i < num_images; ++i)
		images_warped[i].convertTo(images_warped_f[i], CV_32F);

	Ptr<SeamFinder> seam_finder;

	if (seam_find_type_ == "no")
		seam_finder = new detail::NoSeamFinder();
	else if (seam_find_type_ == "voronoi")
		seam_finder = new detail::VoronoiSeamFinder();
	else if (seam_find_type_ == "gc_color")
	{
#ifdef HAVE_OPENCV_GPU
		if (is_try_gpu_ && gpu::getCudaEnabledDeviceCount() > 0)
			seam_finder = new detail::GraphCutSeamFinderGpu(GraphCutSeamFinderBase::COST_COLOR);
		else
#endif
			seam_finder = new detail::GraphCutSeamFinder(GraphCutSeamFinderBase::COST_COLOR);
	}
	else if (seam_find_type_ == "gc_colorgrad")
	{
#ifdef HAVE_OPENCV_GPU
		if (is_try_gpu_ && gpu::getCudaEnabledDeviceCount() > 0)
			seam_finder = new detail::GraphCutSeamFinderGpu(GraphCutSeamFinderBase::COST_COLOR_GRAD);
		else
#endif
			seam_finder = new detail::GraphCutSeamFinder(GraphCutSeamFinderBase::COST_COLOR_GRAD);
	}
	else if (seam_find_type_ == "dp_color")
		seam_finder = new detail::DpSeamFinder(DpSeamFinder::COLOR);
	else if (seam_find_type_ == "dp_colorgrad")
		seam_finder = new detail::DpSeamFinder(DpSeamFinder::COLOR_GRAD);
	if (seam_finder.empty())
	{
		cout << "Can't create the following seam finder '" << seam_find_type_ << "'\n";
		return Stitcher::ERR_NEED_MORE_IMGS;
	}
	seam_finder->find(images_warped_f, corners_, masks_warped);

	images_warped_f.clear();
	return Stitcher::OK;
}

/*
*	�ָ�ԭʼͼ���С
*/
int CosiftStitcher::Rescale(vector<Mat> &src, vector<CameraParams> &cameras, vector<Mat> &seam_masks)
{
	median_focal_len_ = median_focal_len_ / work_scale_;
	Ptr<RotationWarper> warper = warper_creator_->create(median_focal_len_);
	int full_pano_width = cvFloor(median_focal_len_ * 2 * CV_PI);

	//cout << "median focal length: " << median_focal_len_ << endl;

	// Update corners and sizes
	int num_images = static_cast<int>(src.size());
	Mat tmp_mask, tmp_dilated_mask, tmp_seam_mask;
	corners_.clear();
	sizes_.clear();
	for (int src_idx = 0, seam_idx = 0; src_idx < num_images; ++src_idx)
	{
		// Update intrinsics
		cameras[src_idx].focal /= work_scale_;
		cameras[src_idx].ppx /= work_scale_;
		cameras[src_idx].ppy /= work_scale_;

		Mat K;
		cameras[src_idx].K().convertTo(K, CV_32F);

		// ��������image warp������ӳ�����
		Mat tmp_xmap, tmp_ymap;
		warper->buildMaps(src[src_idx].size(), K, cameras[src_idx].R, tmp_xmap, tmp_ymap);

		// Warp the current image mask
		Mat tmp_mask_warped, tmp_final_blend_mask;
		tmp_mask.create(src[src_idx].size(), CV_8U);
		tmp_mask.setTo(Scalar::all(255));
		cv::Point tmp_corner = warper->warp(tmp_mask, K, cameras[src_idx].R, INTER_NEAREST, BORDER_CONSTANT, tmp_mask_warped);

		//	����360��ƴ�ӵ��������
		if (abs(tmp_mask_warped.cols - full_pano_width) <= 10)
		{
			int x1, x2;
			FindWidestInpaintRange(tmp_mask_warped, x1, x2);
			Mat warped_mask[2], blend_mask[2], xmap[2], ymap[2];
			Rect rect[2];
			rect[0] = Rect(0, 0, x1, tmp_mask_warped.rows);
			rect[1] = Rect(x2 + 1, 0, tmp_mask_warped.cols - 1 - x2, tmp_mask_warped.rows);
			for (int j = 0; j < 2; j++)
			{
				tmp_mask_warped(rect[j]).copyTo(warped_mask[j]);
				final_warped_masks_.push_back(warped_mask[j]);

				tmp_xmap(rect[j]).copyTo(xmap[j]);
				xmaps_.push_back(xmap[j]);

				tmp_ymap(rect[j]).copyTo(ymap[j]);
				ymaps_.push_back(ymap[j]);

				// �����ܵ�mask = warp_mask & seam_mask
				dilate(seam_masks[seam_idx], tmp_dilated_mask, Mat());	//����
				resize(tmp_dilated_mask, tmp_seam_mask, rect[j].size());
				final_blend_masks_.push_back(warped_mask[j] & tmp_seam_mask);

				corners_.push_back(tmp_corner + rect[j].tl());
				sizes_.push_back(rect[j].size());

				src_indices_.push_back(src_idx);

				seam_idx++;
			}
		}
		else
		{
			xmaps_.push_back(tmp_xmap);
			ymaps_.push_back(tmp_ymap);
			final_warped_masks_.push_back(tmp_mask_warped);
			corners_.push_back(tmp_corner);

			Size sz = tmp_mask_warped.size();
			sizes_.push_back(sz);

			//	�����ܵ�mask = warp_mask & seam_mask
			dilate(seam_masks[seam_idx], tmp_dilated_mask, Mat());	//����
			resize(tmp_dilated_mask, tmp_seam_mask, sz);
			final_blend_masks_.push_back(tmp_mask_warped & tmp_seam_mask);

			src_indices_.push_back(src_idx);

			seam_idx++;
		}
	}

	dst_roi_ = resultRoi(corners_, sizes_);
	int parts_num = sizes_.size();
	final_warped_images_.resize(parts_num);
	for (int j = 0; j < parts_num; j++)
		final_warped_images_[j].create(sizes_[j], src[src_indices_[j]].type());

	tmp_dilated_mask.release();
	tmp_seam_mask.release();
	tmp_mask.release();

	return Stitcher::OK;
}

/*
*	ƴ�ӽ�������ǲ�������״���ü��ɷ���
*/
int CosiftStitcher::TrimRect(Rect rect)
{
	// ����ÿ��ͼ���rect�����޸�xmap��ymap
	int top = rect.y;
	int left = rect.x;
	int bottom = rect.y + rect.height - 1;
	int right = rect.x + rect.width - 1;
	int num_images = xmaps_.size();
	for (int i = 0; i < num_images; i++)
	{
		int top_i, bottom_i, left_i, right_i;
		top_i = max(dst_roi_.y + top, corners_[i].y);
		left_i = max(dst_roi_.x + left, corners_[i].x);
		bottom_i = min(corners_[i].y + sizes_[i].height - 1, dst_roi_.y + bottom);
		right_i = min(corners_[i].x + sizes_[i].width - 1, dst_roi_.x + right);

		sizes_[i].height = bottom_i - top_i + 1;
		sizes_[i].width = right_i - left_i + 1;

		Rect map_rect(left_i - corners_[i].x, top_i - corners_[i].y,
			sizes_[i].width, sizes_[i].height);

		Mat tmp_map = xmaps_[i].clone();
		tmp_map(map_rect).copyTo(xmaps_[i]);
		tmp_map = ymaps_[i].clone();
		tmp_map(map_rect).copyTo(ymaps_[i]);

		Mat tmp_img = final_blend_masks_[i].clone();
		tmp_img(map_rect).copyTo(final_blend_masks_[i]);

		corners_[i].x = left_i;
		corners_[i].y = top_i;
	}

	dst_roi_.x += left;
	dst_roi_.y += top;
	dst_roi_.width = right - left + 1;
	dst_roi_.height = bottom - top + 1;
	return Stitcher::OK;
}

/*
*	�����ƽ��ͶӰ�Ļ��������Զ�ȥ��δ�������
*/
int CosiftStitcher::TrimInpaint(vector<Mat> &src)
{
	int num_images = static_cast<int>(src.size());

	// �ȼ�������ͼ���mask
	dst_roi_ = resultRoi(corners_, sizes_);
	Mat dst = Mat::zeros(dst_roi_.height, dst_roi_.width, CV_8UC1);
	for (int i = 0; i < num_images; i++)
	{
		int dx = corners_[i].x - dst_roi_.x;
		int dy = corners_[i].y - dst_roi_.y;
		int img_rows = sizes_[i].height;
		int img_cols = sizes_[i].width;
		for (int y = 0; y < img_rows; y++)
		{
			uchar *mask_row_ptr = final_warped_masks_[i].ptr<uchar>(y);
			uchar *dst_row_ptr = dst.ptr<uchar>(dy + y);
			for (int x = 0; x < img_cols; x++)
				dst_row_ptr[dx + x] += mask_row_ptr[x];
		}
	}

	int x, y;
	// top
	for (y = 0; y < dst_roi_.height; y++)
	{
		uchar *dst_row_ptr = dst.ptr<uchar>(y);
		if (!(this->IsRowCrossInpaint(dst_row_ptr, dst_roi_.width)))
			break;
	}
	int top = y;

	// bottom
	for (y = dst_roi_.height - 1; y >= 0; y--)
	{
		uchar *dst_row_ptr = dst.ptr<uchar>(y);
		if (!(this->IsRowCrossInpaint(dst_row_ptr, dst_roi_.width)))
			break;
	}
	int bottom = y;

	// left
	uchar *dst_ptr_00 = dst.ptr<uchar>(0);
	for (x = 0; x < dst_roi_.width; x++)
	{
		for (y = top; y < bottom; y++)
			if (dst_ptr_00[y * (dst_roi_.width) + x] == 0)
				break;
		if (y == bottom)
			break;
	}
	int left = x;

	// right
	for (x = dst_roi_.width - 1; x >= 0; x--)
	{
		for (y = top; y < bottom; y++)
			if (dst_ptr_00[y * (dst_roi_.width) + x] == 0)
				break;
		if (y == bottom)
			break;
	}
	int right = x;

	// ����ÿ��ͼ���rect�����޸�xmap��ymap
	for (int i = 0; i < num_images; i++)
	{
		int top_i, bottom_i, left_i, right_i;
		top_i = max(dst_roi_.y + top, corners_[i].y);
		left_i = max(dst_roi_.x + left, corners_[i].x);
		bottom_i = min(corners_[i].y + sizes_[i].height - 1, dst_roi_.y + bottom);
		right_i = min(corners_[i].x + sizes_[i].width - 1, dst_roi_.x + right);

		sizes_[i].height = bottom_i - top_i + 1;
		sizes_[i].width = right_i - left_i + 1;

		Rect rect(left_i - corners_[i].x, top_i - corners_[i].y,
			sizes_[i].width, sizes_[i].height);

		Mat tmp_map = xmaps_[i].clone();
		tmp_map(rect).copyTo(xmaps_[i]);
		tmp_map = ymaps_[i].clone();
		tmp_map(rect).copyTo(ymaps_[i]);

		Mat tmp_img = final_blend_masks_[i].clone();
		tmp_img(rect).copyTo(final_blend_masks_[i]);

		corners_[i].x = left_i;
		corners_[i].y = top_i;
	}

	dst_roi_.x += left;
	dst_roi_.y += top;
	dst_roi_.width = right - left + 1;
	dst_roi_.height = bottom - top + 1;

	return 0;
}

/*
*	�ж�һ�����Ƿ���δ�������
*/
bool CosiftStitcher::IsRowCrossInpaint(uchar *row, int width)
{
	bool is_have_entered_inpaint = false;
	int count0 = 0;
	for (int x = 1; x < width; x++)
	{
		if (row[x] == 0)
			count0++;
		if (row[x - 1] != 0 && row[x] == 0)
			is_have_entered_inpaint = true;
		if ((row[x - 1] == 0 && row[x] != 0) && is_have_entered_inpaint)
			return true;
	}
	if (count0 >= (width / 2))
		return true;
	return false;
}

int CosiftStitcher::Prepare(vector<Mat> &src, const char* warp_type_)
{
	cv::setBreakOnError(true);
	int num_images = static_cast<int>(src.size());
	if (num_images < 2)
	{
		LOGLN("Need more images");
		return -1;
	}

	//hipDeviceProp_t deviceProp;
	//int deviceCount;
	//hipError_t hipError_t;
	//hipError_t = hipGetDeviceCount(&deviceCount);
	//for (int i = 0; i < deviceCount; i++)
	//{
	//	hipError_t = hipGetDeviceProperties(&deviceProp, i);

	//	cout << "�豸 " << i + 1 << " ����Ҫ���ԣ� " << endl;
	//	cout << "�豸�Կ��ͺţ� " << deviceProp.name << endl;
	//	cout << "�豸ȫ���ڴ���������MBΪ��λ���� " << deviceProp.totalGlobalMem / 1024 / 1024 << endl;
	//	cout << "�豸��һ���߳̿飨Block���п��õ�������ڴ棨��KBΪ��λ���� " << deviceProp.sharedMemPerBlock / 1024 << endl;
	//	cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << deviceProp.regsPerBlock << endl;
	//	cout << "�豸��һ���߳̿飨Block���ɰ���������߳������� " << deviceProp.maxThreadsPerBlock << endl;
	//	cout << "�豸�ļ��㹦�ܼ���Compute Capability���İ汾�ţ� " << deviceProp.major << "." << deviceProp.minor << endl;
	//	cout << "�豸�϶ദ������������ " << deviceProp.multiProcessorCount << endl;
	//}

	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != 0)
	{
		LOGLN("GPU acceleration failed! Error code: " << cudaStatus << " Please ensure that you have a CUDA-capable GPU installed!");
		LOGLN("Stitching with CPU next ...");
		return -1;
	}

	hipError_t flag;
	if (warp_type_ == "apap")
		flag = PrepareAPAP(src);
	else
		flag = PrepareClassical(src);

		if (flag == hipSuccess)
		{
			flag = DevMalloc(num_images);
			if (flag != hipSuccess)
				return flag;
			C2GInitData *c2g_data = new C2GInitData[num_images];
			for (int i = 0; i < num_images; i++)
			{
				c2g_data[i].xmap = xmaps_[i].ptr<float>(0);
				c2g_data[i].ymap = ymaps_[i].ptr<float>(0);
				c2g_data[i].ec_weight = ec_weight_maps_[i].ptr<float>(0);
				c2g_data[i].blend_weight = blend_weight_maps_[i].ptr<float>(0);
				c2g_data[i].total_weight = total_weight_maps_[i].ptr<float>(0);
				c2g_data[i].height = src[i].rows;
				c2g_data[i].width = src[i].cols;
				c2g_data[i].warped_height = xmaps_[i].rows;
				c2g_data[i].warped_width = xmaps_[i].cols;
				c2g_data[i].corner_x = corners_[i].x - dst_roi_.x;
				c2g_data[i].corner_y = corners_[i].y - dst_roi_.y;
			}
			DevDataUpload(c2g_data, dst_roi_.height, dst_roi_.width);
		}

	if (flag == hipSuccess)
	{
		LOGLN("\t~Prepare complete");
		is_prepared_ = true;
	}

	return flag;
}

/*
*	APAP�㷨�ĳ�ʼ��
*/
hipError_t CosiftStitcher::PrepareAPAP(vector<Mat> &src)
{
	int num_images = static_cast<int>(src.size());

	this->InitMembers(num_images);

	// ����һЩ�����ĳ߶ȣ����������ͼ���ӷ��ʱ��Ϊ����߳���Ч�ʣ����Զ�Դͼ�����һЩ����
	work_megapix_ = -1;	//	�Ȳ����Ƿ���
	seam_megapix_ = -1;	//	�Ȳ����Ƿ���
	this->SetScales(src);

	// �������
	vector<ImageFeatures> features(num_images);
	this->FindFeatures(src, features);

	// ����ƥ�䣬��ȥ������ͼƬ
	vector<MatchesInfo> pairwise_matches;
	this->MatchImages(features, pairwise_matches);

	// APAP�㷨
	//APAPWarper apap_warper;
	//apap_warper.buildMaps(src, features, pairwise_matches, xmaps_, ymaps_, corners_);
	for (int i = 0; i < num_images; i++)
		sizes_[i] = xmaps_[i].size();
	dst_roi_ = resultRoi(corners_, sizes_);

	// ����ӷ�
	vector<Mat> seamed_masks(num_images);
	vector<Mat> images_warped(num_images);
	vector<Mat> init_masks(num_images);
	for (int i = 0; i < num_images; i++)
	{
		init_masks[i].create(src[i].size(), CV_8U);
		init_masks[i].setTo(Scalar::all(255));
		remap(src[i], images_warped[i], xmaps_[i], ymaps_[i], INTER_LINEAR);
		remap(init_masks[i], final_warped_masks_[i], xmaps_[i], ymaps_[i], INTER_NEAREST, BORDER_CONSTANT);
		seamed_masks[i] = final_warped_masks_[i].clone();
	}
	this->FindSeam(images_warped, seamed_masks);
	LOGLN("find seam");

	// �عⲹ��
	compensator_.createWeightMaps(corners_, images_warped, final_warped_masks_, ec_weight_maps_);
	// �عⲹ��ʱ��������ȨֵҲҪresizeһ��
	compensator_.gainMapResize(sizes_, ec_weight_maps_);
	LOGLN("compensate");

	images_warped.clear();

	// �����ں�ʱ�������ص�Ȩֵ
	Size dst_sz = dst_roi_.size();
	//cout << "dst size: " << dst_sz << endl;
	float blend_width = sqrt(static_cast<float>(dst_sz.area())) * blend_strength_ / 100.f;
	blender_.setSharpness(1.f / blend_width);
	for (int i = 0; i < num_images; i++)
		final_blend_masks_[i] = final_warped_masks_[i] & seamed_masks[i];
	blender_.createWeightMaps(dst_roi_, corners_, seamed_masks, blend_weight_maps_);

	return hipError_t::hipSuccess;
}

/*
*	Classical�㷨�ĳ�ʼ��
*/
hipError_t CosiftStitcher::PrepareClassical(vector<Mat> &src)
{
	int num_images = static_cast<int>(src.size());
	LOGLN("Preparing...");

	this->InitMembers(num_images);

	// ����һЩ�����ĳ߶ȣ����������ͼ���ӷ��ʱ��Ϊ����߳���Ч�ʣ����Զ�Դͼ�����һЩ����
	this->SetScales(src);

	if ((cameras_.size() == 0) || (cameras_.size() != num_images))
	{
		if ((cameras_.size() != 0) && (cameras_.size() != num_images))
		{
			cameras_.clear();
			LOGLN("\t~load camera parameters error! Trying to calculate again ...");
		}

		// �������
		LOGLN("\t~finding features...");
		vector<ImageFeatures> features(num_images);
		this->FindFeatures(src, features);

		// ����ƥ�䣬��ȥ������ͼƬ
		LOGLN("\t~matching images...");
		vector<MatchesInfo> pairwise_matches;
		int retrun_flag = this->MatchImages(features, pairwise_matches);
		if (retrun_flag != 0)
			return hipError_t::hipErrorNotReady;

		// ������궨
		LOGLN("\t~calibrating cameras...");
		cameras_.resize(num_images);
		this->CalibrateCameras(features, pairwise_matches, cameras_);
	}


	//	����ˮƽ�ӽǣ��ж�ƽ��ͶӰ�ĺϷ���
	LOGLN("\t~calculating view angle...");
	view_angle_ = this->GetViewAngle(src, cameras_);
	if (view_angle_ > 140 && warp_type_ == "plane")
		warp_type_ = "cylindrical";

	// Ϊ�ӷ�ļ�����Warp
	LOGLN("\t~warping for seaming...");
	vector<Mat> masks_warped;
	vector<Mat> images_warped;
	this->WarpForSeam(src, cameras_, masks_warped, images_warped);

	// �عⲹ��
	LOGLN("\t~compensating...");
	compensator_.createWeightMaps(corners_, images_warped, masks_warped, ec_weight_maps_);

	// ����ӷ�
	LOGLN("\t~finding seam...");
	this->FindSeam(images_warped, masks_warped);
	images_warped.clear();

	// �������������masks��ԭ��������С
	LOGLN("\t~rescaling...");
	this->Rescale(src, cameras_, masks_warped);

	// �ü���inpaint����
	if (trim_type_ == CosiftStitcher::TRIM_AUTO)
		if (warp_type_ == "plane")
			this->TrimInpaint(src);
	if (trim_type_ == CosiftStitcher::TRIM_RECTANGLE)
		this->TrimRect(trim_rect_);

	// ƴ������
	//this->RegistEvaluation(features, pairwise_matches, cameras);

	// �عⲹ��ʱ��������ȨֵҲҪresizeһ��
	LOGLN("\t~resizing compensators' weight map...");
	compensator_.gainMapResize(sizes_, ec_weight_maps_);

	// �����ں�ʱ�������ص�Ȩֵ
	LOGLN("\t~blending...");
	Size dst_sz = dst_roi_.size();
	float blend_width = sqrt(static_cast<float>(dst_sz.area())) * blend_strength_ / 100.f;
	blender_.setSharpness(1.f / blend_width);
	blender_.createWeightMaps(dst_roi_, corners_, final_blend_masks_, blend_weight_maps_);

	//	������Ȩ��
	num_images = sizes_.size();
	total_weight_maps_.resize(num_images);
	for (int i = 0; i < num_images; i++)
	{
		int n_pixel = sizes_[i].height * sizes_[i].width;
		float *blend_weight_ptr = blend_weight_maps_[i].ptr<float>(0);
		float *ec_weight_ptr = ec_weight_maps_[i].ptr<float>(0);
		total_weight_maps_[i].create(sizes_[i]);
		float *total_weight_ptr = total_weight_maps_[i].ptr<float>(0);
		for (int j = 0; j < n_pixel; j++)
			total_weight_ptr[j] = blend_weight_ptr[j] * ec_weight_ptr[j];
	}
	//	����xmap��ymap������GPU�˺���ʹ��
	for (int i = 0; i < num_images; i++)
	{
		float *xmap = xmaps_[i].ptr<float>(0);
		float *ymap = ymaps_[i].ptr<float>(0);
		int n_pixel = sizes_[i].height * sizes_[i].width;
		int src_height = src[src_indices_[i]].rows;
		int src_width = src[src_indices_[i]].cols;
		for (int j = 0; j < n_pixel; j++)
		{
			float map_x = xmap[j];
			float map_y = ymap[j];
			int map_x1 = cvFloor(map_x);
			int map_y1 = cvFloor(map_y);
			int map_x2 = map_x1 + 1;
			int map_y2 = map_y1 + 1;
			if ((map_x1 < 0) || (map_y1 < 0) || (map_x2 >= src_width) || (map_y2 >= src_height))
				xmap[j] = ymap[j] = -1;
		}
	}

	//is_prepared_ = true;
	return hipError_t::hipSuccess;
}

int CosiftStitcher::StitchFrame(vector<Mat> &src, Mat &dst)
{
	if (!is_prepared_)
	{
		int flag = Prepare(src);
		if (flag != 0)
			return flag;
	}

	if (is_try_gpu_)
		return StitchFrameGPU(src, dst);
	else
		return StitchFrameCPU(src, dst);
}

int CosiftStitcher::StitchFrameGPU(vector<Mat> &src, Mat &dst)
{
	if (dst.empty())
		dst.create(dst_roi_.size(), CV_8UC3);

	int image_num = src.size();
	GPUImageData *images = new GPUImageData[image_num];
	for (int i = 0; i < image_num; i++)
		images[i].data = src[i].ptr<uchar>(0);
	int flag = Cuda_Stitch(images, dst.ptr<uchar>(0));
	free(images);
	return flag;
}

int CosiftStitcher::StitchFrameCPU(vector<Mat> &src, Mat &dst)
{
	bool time_debug = false;//true;//
	long start_clock, end_clock;

	if (time_debug)
		start_clock = clock();

	int64 t;
	int num_images = src_indices_.size();

	int dst_width = dst_roi_.width;
	int dst_height = dst_roi_.height;
	if (dst.empty())
		dst.create(dst_roi_.size(), CV_8UC3);
	uchar *dst_ptr_00 = dst.ptr<uchar>(0);
	memset(dst_ptr_00, 0, dst_width * dst_height * 3);

	double warp_time[100], feed_time[100];

	for (int img_idx = 0; img_idx < num_images; ++img_idx)
	{
		if (time_debug)
			t = getTickCount();

		// Warp the current image
		remap(src[src_indices_[img_idx]], final_warped_images_[img_idx], xmaps_[img_idx], ymaps_[img_idx],
			INTER_LINEAR);//, BORDER_REFLECT);
		if (time_debug)
			warp_time[img_idx] = 1000 * (getTickCount() - t) / getTickFrequency();

		if (time_debug)
			t = getTickCount();
		int dx = corners_[img_idx].x - dst_roi_.x;
		int dy = corners_[img_idx].y - dst_roi_.y;
		int img_rows = sizes_[img_idx].height;
		int img_cols = sizes_[img_idx].width;
		int src_rows = src[img_idx].rows;
		int src_cols = src[img_idx].cols;

		int rows_per_parallel = img_rows / parallel_num_;
#pragma omp parallel for
		for (int parallel_idx = 0; parallel_idx < parallel_num_; parallel_idx++)
		{
			int row_start = parallel_idx * rows_per_parallel;
			int row_end = row_start + rows_per_parallel;
			if (parallel_idx == parallel_num_ - 1)
				row_end = img_rows;

			uchar *dst_ptr;
			uchar *warped_img_ptr = final_warped_images_[img_idx].ptr<uchar>(row_start);
			float *total_weight_ptr = total_weight_maps_[img_idx].ptr<float>(row_start);
			for (int y = row_start; y < row_end; y++)
			{
				dst_ptr = dst_ptr_00 + ((dy + y) * dst_width + dx) * 3;
				for (int x = 0; x < img_cols; x++)
				{
					/* �عⲹ�����ںϼ�Ȩƽ�� */
					(*dst_ptr) += (uchar)(cvRound((*warped_img_ptr) * (*total_weight_ptr)));
					warped_img_ptr++;
					dst_ptr++;

					(*dst_ptr) += (uchar)(cvRound((*warped_img_ptr) * (*total_weight_ptr)));
					warped_img_ptr++;
					dst_ptr++;

					(*dst_ptr) += (uchar)(cvRound((*warped_img_ptr) * (*total_weight_ptr)));
					warped_img_ptr++;
					dst_ptr++;

					total_weight_ptr++;
				}
			}
		}


		if (time_debug)
			feed_time[img_idx] = 1000 * (getTickCount() - t) / getTickFrequency();
	}

	if (time_debug)
		for (int i = 0; i < num_images; i++)
			cout << "\twarp " << warp_time[i] << "ms, feed " << feed_time[i] << "ms" << endl;

	if (time_debug)
		cout << "(=" << clock() - start_clock << "ms)" << endl;

	return 0;
}

void CosiftStitcher::setDebugDirPath(string dir_path)
{
	is_debug_ = true;
	debug_dir_path_ = dir_path;
}

int CosiftStitcher::RegistEvaluation(vector<ImageFeatures> &features, vector<MatchesInfo> &pairwise_matches, vector<CameraParams> &cameras)
{
	int num_images = features.size();
	Ptr<RotationWarper> warper = warper_creator_->create(median_focal_len_);

	MatchesInfo matches_info;
	vector<vector<cv::Point2f>> warped_fpts;
	warped_fpts.resize(num_images);
	for (int i = 0; i < num_images; i++)
	{
		int fpts_num = features[i].keypoints.size();
		warped_fpts[i].resize(fpts_num);
		Mat K;
		cameras[i].K().convertTo(K, CV_32F);
		for (int j = 0; j < fpts_num; j++)
			warped_fpts[i][j] = warper->warpPoint(features[i].keypoints[j].pt, K, cameras[i].R);
	}

	double final_total_error, final_total_inliners;
	final_total_inliners = final_total_error = 0;

	for (int i = 0; i < num_images; i++)
	{
		for (int j = i + 1; j < num_images; j++)
		{
			// �������
			int idx = i * num_images + j;
			matches_info = pairwise_matches[idx];

			int inliner_nums = matches_info.num_inliers;
			if (inliner_nums < 50)// || j != i+1)
				continue;

			int matches_size = matches_info.matches.size();
			double total_error = 0;
			for (int k = 0; k < matches_size; k++)
			{
				if (matches_info.inliers_mask[k])
				{
					const DMatch& m = matches_info.matches[k];
					cv::Point2f p1 = warped_fpts[i][m.queryIdx];
					cv::Point2f p2 = warped_fpts[j][m.trainIdx];
					total_error += ((p1.x - p2.x) * (p1.x - p2.x) + (p1.y - p2.y) * (p1.y - p2.y));
				}
			}
			final_total_error += total_error;
			final_total_inliners += inliner_nums;
			LOGLN("\t\t~Image" << i << "-" << j << ": total error(" << total_error <<
				"), total inliners(" << inliner_nums << "), average error(" <<
				sqrt(total_error / inliner_nums) << ")");
		}
	}
	LOGLN("\t\t~all pairs' total error(" << final_total_error <<
		"), total inliners(" << final_total_inliners << "), average error(" <<
		sqrt(final_total_error / final_total_inliners) << ")");

	return 0;
}

int CosiftStitcher::stitchImage(vector<Mat> &src, Mat &pano)
{
	Prepare(src);
	if (false)
	{
		char img_name[100];
		int img_num = corners_.size();
		cout << dst_roi_ << endl;
		for (int i = 0; i < img_num; i++)
		{
			cout << src_indices_[i] << ", " << corners_[i] << ", " << sizes_[i] << endl;
			sprintf(img_name, "/masks/%d.jpg", i);
			imwrite(debug_dir_path_ + img_name, this->final_blend_masks_[i]);

			sprintf(img_name, "/weight/%d.jpg", i);
			Mat weight_img_float = total_weight_maps_[i] * 255;
			Mat weight_img;
			weight_img_float.convertTo(weight_img, CV_8U);
			imwrite(debug_dir_path_ + img_name, weight_img);
		}
	}
	StitchFrame(src, pano);
	return 0;
}

//	����������������ļ���ʽ���£�
//	��һ�����м佹��median_focal_len_
//	֮��ÿһ����һ�����--
//		����������focal��aspect��ppx��ppy��R��t
void CosiftStitcher::saveCameraParam(string filename)
{
	ofstream cp_file(filename.c_str());
	cp_file << median_focal_len_ << endl;
	for (int i = 0; i < cameras_.size(); i++)
	{
		CameraParams cp = cameras_[i];
		cp_file << cp.focal << " " << cp.aspect << " " << cp.ppx << " " << cp.ppy;
		for (int r = 0; r < 3; r++)
			for (int c = 0; c < 3; c++)
				cp_file << " " << cp.R.at<float>(r, c);
		for (int r = 0; r < 3; r++)
			cp_file << " " << cp.t.at<double>(r, 0);
		cp_file << endl;
	}
	cp_file.close();
}

int CosiftStitcher::loadCameraParam(string filename)
{
	ifstream cp_file(filename.c_str());
	string line;

	//	median_focal_len_
	if (!getline(cp_file, line))
		return -1;
	stringstream mfl_string_stream;
	mfl_string_stream << line;
	mfl_string_stream >> median_focal_len_;

	//	ÿ��һ�������
	cameras_.clear();
	while (getline(cp_file, line))
	{
		stringstream cp_string_stream;
		cp_string_stream << line;
		CameraParams cp;
		cp.R.create(3, 3, CV_32F);
		cp.t.create(3, 1, CV_64F);
		cp_string_stream >> cp.focal >> cp.aspect >> cp.ppx >> cp.ppy;
		for (int r = 0; r < 3; r++)
			for (int c = 0; c < 3; c++)
				cp_string_stream >> cp.R.at<float>(r, c);
		for (int r = 0; r < 3; r++)
			cp_string_stream >> cp.t.at<double>(r, 0);
		cameras_.push_back(cp);
	}
	return 0;
}